#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>

// using global memory coalescing
// concept of warp and warp scheduler
// while loading from global memory, warp scheduler can load 4 consec memory locations in single transaction

// we are trying to minimize the loads (instead of sepearte loads for every thread)
// Now just take care what u access in the function

__global__
void
sgemm_naive(int M, int N, int K, float alpha, const float *A, const float *B, float beta, float *C)
{
    const int x = blockIdx.x * 32 + (threadIdx.x / 32);
    const int y = blockIdx.y * 32 + (threadIdx.x % 32);

    if(x < M && y < M) {
        float tmp = 0.0;
        for(int i=0; i<K; i++){
            tmp += A[x * K + i] * B[i * N + y];
        }

        C[x * N + y] = alpha * tmp + beta * C[x * N + y];
    }
}

int
main()
{
    int n = 1 << 10;

    size_t bytes = n * n *  sizeof(float);
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT); 


    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

    //filling with random numbers

    hiprandGenerateUniform(prng, d_a, n*n);
    hiprandGenerateUniform(prng, d_b, n*n);

    const int M = 1024;
    const int N = 1024;
    const int K = 1024;

    dim3 gridDim(32,32,1);
    dim3 blockDim(32,32,1);
    
    clock_t start = clock();
    sgemm_naive<<<gridDim,blockDim>>>(M,N,K,1.0f,d_a,d_b,0.0f,d_c);
    
    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
    clock_t end = clock();

    double cuda_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
    printf("GMEMcol Execution time: %f seconds\n", cuda_time_used);
} 